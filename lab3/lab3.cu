#include "hip/hip_runtime.h"
#include <wb.h>
#define BLOCK_SIZE 32
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float subTileB[BLOCK_SIZE][BLOCK_SIZE];
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  float Pvalue=0;

  for (int m = 0; m < (numAColumns-1)/BLOCK_SIZE+1; m++) {
    if (BLOCK_SIZE*m + tx < numAColumns) {
      subTileA[ty][tx] = A[numAColumns*row + BLOCK_SIZE*m + tx];
    }
    else {
      subTileA[ty][tx] = 0.0f;
    }

    if (BLOCK_SIZE*m + ty < numBRows) {
      subTileB[ty][tx] = B[(BLOCK_SIZE*m+ty)*numBColumns + col];
    }
    else {
      subTileB[ty][tx] = 0.0f;
    }

    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++)
      if (row < numARows && col < numBColumns)
        Pvalue += subTileA[ty][k] * subTileB[k][tx];

    __syncthreads();

  }

  if (row < numARows && col < numBColumns)
    C[row*numBColumns+col] = Pvalue;

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  float *devA;
  float *devB;
  float *devC;

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  wbTime_stop(Generic, "Importing data and creating memory on host");
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  //@@ Allocate GPU memory here
  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **)&devA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&devB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&devC, numCRows * numCColumns * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");


  //@@ Copy memory to the GPU here
  wbTime_start(GPU, "Copying input to the GPU.");
  hipMemcpy(devA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  wbTime_start(Compute, "Performing CUDA computation");


  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float)numCColumns/BLOCK_SIZE), ceil((float)numCRows/BLOCK_SIZE),1);
  dim3 DimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
  

  //@@ Launch the GPU Kernel here
  wbTime_start(Compute, "Performing CUDA computation");
  matrixMultiplyShared<<<DimGrid,DimBlock>>>(devA, devB, devC, numARows, numAColumns, numBRows,
      numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");
  //@@ Copy the GPU memory back to the CPU here
  wbTime_stop(Copy, "Copying output to the CPU");
  hipMemcpy(hostC, devC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output to the CPU");

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Free GPU Memory");
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  wbTime_start(GPU, "Free GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);
  //@@ Free the hostC matrix

  return 0;
}
