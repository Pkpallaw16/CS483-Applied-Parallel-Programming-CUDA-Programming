#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
//m3_op2.out
// Apply __restrict__ to the pointer parameters
__global__ void matrix_unrolling_kernel(const float *__restrict__ input, float *__restrict__ output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]

    int b = blockIdx.z;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;

    const int H_out = Height - K + 1;
    const int W_out = Width - K + 1;
    const int W_unroll = H_out * W_out;
    const int H_unroll = Channel * K * K;

    if (h_out < H_out && w_out < W_out && b < Batch) {
        int w_unroll = h_out * W_out + w_out;

        for (int c = 0; c < Channel; c++) {
            int w_base = c * K * K;
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    int h_unroll = w_base + p * K + q;
                    float val = in_4d(b, c, h_out + p, w_out + q);
                    size_t indx = ((size_t)h_unroll) * (Batch * W_unroll) + b * W_unroll + w_unroll;
                    output[indx] = val;
                }
            }
        }
    }
    #undef in_4d
}

__global__ void matrixMultiplyShared(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t)row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t)tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

__global__ void matrix_permute_kernel(const float *__restrict__ input, float *__restrict__ output,
                                      int Map_out, int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input,
                                                    const float *host_mask, float **device_output_ptr,
                                                    float **device_input_ptr, float **device_mask_ptr,
                                                    const int Batch, const int Map_out, const int Channel,
                                                    const int Height, const int Width, const int K) {
    size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    size_t output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float);
    size_t mask_size = Map_out * Channel * K * K * sizeof(float);

    // Allocate device memory
    hipMalloc((void **) device_input_ptr, input_size);
    hipMalloc((void **) device_output_ptr, output_size);
    hipMalloc((void **) device_mask_ptr, mask_size);

    // Copy data from host to device
    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);
}

__host__ void GPUInterface::conv_forward_gpu(float * __restrict__ device_output, const float * __restrict__ device_input,
                                             const float * __restrict__ device_mask, const int Batch,
                                             const int Map_out, const int Channel,
                                             const int Height, const int Width, const int K) {
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int H_unroll = Channel * K * K;
    const int W_unroll = Height_out * Width_out;

    float *__restrict__ unrolled_matrix;
    float *__restrict__ matmul_output;

    size_t unrolled_size = (size_t)H_unroll * Batch * W_unroll * sizeof(float);
    size_t matmul_output_size = (size_t)Map_out * Batch * W_unroll * sizeof(float);

    hipMalloc((void **) &unrolled_matrix, unrolled_size);
    hipMalloc((void **) &matmul_output, matmul_output_size);

    // Kernel dimensions for unrolling
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim((Width_out + TILE_WIDTH - 1) / TILE_WIDTH,
                 (Height_out + TILE_WIDTH - 1) / TILE_WIDTH,
                 Batch);

    matrix_unrolling_kernel<<<gridDim, blockDim>>>(device_input, unrolled_matrix,
                                                   Batch, Channel, Height, Width, K);

    // Matrix multiplication dimensions
    int numARows = Map_out;
    int numAColumns = H_unroll;
    int numBRows = H_unroll;
    int numBColumns = Batch * W_unroll;
    int numCRows = Map_out;
    int numCColumns = Batch * W_unroll;

    dim3 gridDimMatMul((numCColumns + TILE_WIDTH - 1) / TILE_WIDTH,
                       (numCRows + TILE_WIDTH - 1) / TILE_WIDTH);

    matrixMultiplyShared<<<gridDimMatMul, blockDim>>>(device_mask, unrolled_matrix,
                                                      matmul_output, numARows, numAColumns,
                                                      numBRows, numBColumns,
                                                      numCRows, numCColumns);

    // Permute the result of matrix multiplication
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);

    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(matmul_output,
                                                                   device_output,
                                                                   Map_out, Batch,
                                                                   out_image_size);

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output,
                                                    float *device_input, float *device_mask,
                                                    const int Batch, const int Map_out, const int Channel,
                                                    const int Height, const int Width, const int K) {
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMemcpy(host_output, device_output, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
        std::cout << "Computational capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
        std::cout << "Max Constant memory size: " << deviceProp.totalConstMem << std::endl;
        std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;
        std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, "
                  << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2] << " z" << std::endl;
        std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, "
                  << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2] << " z" << std::endl;
        std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
    }
}

